#define DOCTEST_CONFIG_IMPLEMENT_WITH_MAIN

#include <doctest.h>
#include <taskflow/taskflow.hpp>
#include <taskflow/cuda/cudaflow.hpp>

TEST_CASE("cuda.version" * doctest::timeout(300) ) {
  REQUIRE(tf::cuda_get_driver_version() > 0);
  REQUIRE(tf::cuda_get_runtime_version() > 0);
}

TEST_CASE("cuda.device" * doctest::timeout(300) ) {

  REQUIRE(tf::cuda_get_num_devices() > 0);
  REQUIRE(tf::cuda_get_device() >= 0);

  size_t num_devices = tf::cuda_get_num_devices();

  for(size_t d=0; d<num_devices; d++) {
    tf::cuda_set_device(d);
    REQUIRE(tf::cuda_get_device() == d);
    
    for(size_t d=0; d<num_devices; d++) {
      REQUIRE(tf::cuda_get_device_max_threads_per_block(d) > 0);
      REQUIRE(tf::cuda_get_device_max_x_dim_per_block(d) > 0);
      REQUIRE(tf::cuda_get_device_max_y_dim_per_block(d) > 0);
      REQUIRE(tf::cuda_get_device_max_z_dim_per_block(d) > 0);
      REQUIRE(tf::cuda_get_device_max_x_dim_per_grid(d) > 0);
      REQUIRE(tf::cuda_get_device_max_y_dim_per_grid(d) > 0);
      REQUIRE(tf::cuda_get_device_max_z_dim_per_grid(d) > 0);
      REQUIRE(tf::cuda_get_device_warp_size(d) > 0);
      REQUIRE(tf::cuda_get_device_max_shm_per_block(d) > 0);
      REQUIRE(tf::cuda_get_device_compute_capability_major(d) > 0);
      REQUIRE(tf::cuda_get_device_compute_capability_minor(d) >= 0);
      REQUIRE_NOTHROW(tf::cuda_get_device_unified_addressing(d));
    }
  }
  
  // going back to device 0
  tf::cuda_set_device(0);
}

// ----------------------------------------------------------------------------
// stream
// ----------------------------------------------------------------------------

TEST_CASE("cudaStream" * doctest::timeout(300)) {
  
  // create a new stream s1 inside
  tf::cudaStream s1;
  
  // create another stream s2 from the outside
  hipStream_t s2_source;
  hipStreamCreate(&s2_source);
  tf::cudaStream s2(s2_source);
  
  REQUIRE(s2 == s2_source);

  hipStream_t s1_source = s1;
  REQUIRE(s1 == s1_source);

  // query status
  REQUIRE(hipStreamQuery(s1) == hipSuccess);
  REQUIRE(hipStreamQuery(s2) == hipSuccess);

  s1 = std::move(s2);

  REQUIRE(s2 == nullptr);
  REQUIRE(s1 == s2_source);
  REQUIRE(hipStreamQuery(s1) == hipSuccess);

}

// ----------------------------------------------------------------------------
// event
// ----------------------------------------------------------------------------

TEST_CASE("cudaEvent" * doctest::timeout(300)) {
  
  // create a new event e1 inside
  tf::cudaEvent e1;
  
  // create another event e2 from the outside
  hipEvent_t e2_source;
  hipEventCreate(&e2_source);
  tf::cudaEvent e2(e2_source);
  
  REQUIRE(e2 == e2_source);

  hipEvent_t e1_source = e1;
  REQUIRE(e1 == e1_source);

  // query status
  REQUIRE(hipEventQuery(e1) == hipSuccess);
  REQUIRE(hipEventQuery(e2) == hipSuccess);

  e1 = std::move(e2);

  REQUIRE(e2 == nullptr);
  REQUIRE(e1 == e2_source);
  REQUIRE(hipEventQuery(e1) == hipSuccess);
  REQUIRE(hipEventQuery(e2) != hipSuccess);
}

