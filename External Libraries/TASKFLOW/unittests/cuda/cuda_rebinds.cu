#include "hip/hip_runtime.h"
#define DOCTEST_CONFIG_IMPLEMENT_WITH_MAIN

#include <doctest.h>

#include <taskflow/taskflow.hpp>
#include <taskflow/cuda/cudaflow.hpp>
#include <taskflow/cuda/algorithm/reduce.hpp>
#include <taskflow/cuda/algorithm/for_each.hpp>
#include <taskflow/cuda/algorithm/transform.hpp>

//verify
template <typename T>
__global__
void verify(const T* a, const T* b, bool* check, size_t size) {
  size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  for(;tid < size; tid += gridDim.x * blockDim.x) {
    if(a[tid] != b[tid]) {
      *check = false;
      return;
    }
  }
}

//add
template <typename T>
__global__
void add(const T* a, const T* b, T* c, size_t size) {
  size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  for(;tid < size; tid += gridDim.x * blockDim.x) {
    c[tid] = a[tid] + b[tid];
  }
}

//multiply
template <typename T>
__global__
void multiply(const T* a, const T* b, T* c, size_t size) {
  size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  for(;tid < size; tid += gridDim.x * blockDim.x) {
    c[tid] = a[tid] * b[tid];
  }
}

//----------------------------------------------------------------------
//rebind kernel
//----------------------------------------------------------------------

template <typename T>
void rebind_kernel() {
  tf::Executor executor;

  for(size_t N = 1; N < 65529; N = N * 2 + 1) {
    tf::Taskflow taskflow;

    std::vector<T*> operand(3, nullptr);
    std::vector<T*> ans_operand(3, nullptr);

    std::vector<int> ind(3);
    std::generate_n(ind.data(), 3, [&](){ return ::rand() % 3; });


    bool* check {nullptr};

    //allocate
    auto allocate_t = taskflow.emplace([&]() {
      for(int i = 0; i < 3; ++i) {
        REQUIRE(hipMallocManaged(&operand[i], N * sizeof(T)) == hipSuccess);
        REQUIRE(hipMallocManaged(&ans_operand[i], N * sizeof(T)) == hipSuccess);
      }

      REQUIRE(hipMallocManaged(&check, sizeof(bool)) == hipSuccess);
    }).name("allocate");

    //initialize
    auto initialize_t = taskflow.emplace([&](){
      for(int i = 0; i < 3; ++i) {
        std::generate_n(operand[i], N, [&](){ return ::rand() % N - N / 2 + i; });
        std::memcpy(ans_operand[i], operand[i], N * sizeof(T));
      }
      
      *check = true;
    }).name("initialize"); 

    
    //rebind_kernel
    auto add_t = taskflow.emplace([&](tf::cudaFlowCapturer& cf) {
      auto multi_t = cf.kernel(
        32, 512, 0,
        multiply<T>,
        operand[ind[0]], operand[ind[1]], operand[ind[2]], N
      );

      auto add_t = cf.kernel(
        32, 512, 0,
        add<T>,
        operand[ind[1]], operand[ind[2]], operand[ind[0]], N
      );

      multi_t.precede(add_t);

      cf.offload();

      cf.kernel(
        multi_t,
        64, 128, 0,
        multiply<T>,
        operand[ind[2]], operand[ind[0]], operand[ind[1]], N
      );

      cf.kernel(
        add_t,
        16, 256, 0,
        add<T>,
        operand[ind[1]], operand[ind[0]], operand[ind[2]], N
      );

      cf.offload();

      cf.kernel(
        multi_t,
        8, 1024, 0,
        multiply<T>,
        operand[ind[0]], operand[ind[2]], operand[ind[1]], N
      );

      cf.kernel(
        add_t,
        64, 64, 0,
        add<T>,
        operand[ind[2]], operand[ind[1]], operand[ind[0]], N
      );

      cf.offload();
    }).name("add");

    //verify
    auto verify_t = taskflow.emplace([&](tf::cudaFlowCapturer& cf) {
      //auto multi1_t = cf.transform(
      //  ans_operand[ind[2]],  ans_operand[ind[2]]+ N,
      //  [] __device__ (T& v1, T& v2) { return v1 * v2; },
      //  ans_operand[ind[0]], ans_operand[ind[1]]
      //);

      auto multi1_t = cf.transform(
        ans_operand[ind[0]], ans_operand[ind[0]] + N, ans_operand[ind[1]],
        ans_operand[ind[2]],
        [] __device__ (T& v1, T& v2) { return v1*v2; }
      );

      //auto add1_t = cf.transform(
      //  ans_operand[ind[0]],  ans_operand[ind[0]]+ N,
      //  [] __device__ (T& v1, T& v2) { return v1 + v2; },
      //  ans_operand[ind[1]], ans_operand[ind[2]]
      //);

      auto add1_t = cf.transform(
        ans_operand[ind[1]], ans_operand[ind[1]]+N, ans_operand[ind[2]],
        ans_operand[ind[0]],
        [] __device__ (T& v1, T& v2) { return v1 + v2; }
      );

      //auto multi2_t = cf.transform(
      //  ans_operand[ind[1]],  ans_operand[ind[1]]+ N,
      //  [] __device__ (T& v1, T& v2) { return v1 * v2; },
      //  ans_operand[ind[2]], ans_operand[ind[0]]
      //);
      
      auto multi2_t = cf.transform(
        ans_operand[ind[2]], ans_operand[ind[2]] + N, ans_operand[ind[0]],
        ans_operand[ind[1]],
        [] __device__ (T& v1, T& v2) { return v1 * v2; }
      );

      //auto add2_t = cf.transform(
      //  ans_operand[ind[2]],  ans_operand[ind[2]]+ N,
      //  [] __device__ (T& v1, T& v2) { return v1 + v2; },
      //  ans_operand[ind[1]], ans_operand[ind[0]]
      //);
      
      auto add2_t = cf.transform(
        ans_operand[ind[1]], ans_operand[ind[1]] + N, ans_operand[ind[0]],
        ans_operand[ind[2]],
        [] __device__ (T& v1, T& v2) { return v1 + v2; }
      );

      //auto multi3_t = cf.transform(
      //  ans_operand[ind[1]],  ans_operand[ind[1]]+ N,
      //  [] __device__ (T& v1, T& v2) { return v1 * v2; },
      //  ans_operand[ind[0]], ans_operand[ind[2]]
      //);
      
      auto multi3_t = cf.transform(
        ans_operand[ind[0]], ans_operand[ind[0]] + N,  ans_operand[ind[2]],
        ans_operand[ind[1]],
        [] __device__ (T& v1, T& v2) { return v1 * v2; }
      );

      //auto add3_t = cf.transform(
      //  ans_operand[ind[0]],  ans_operand[ind[0]]+ N,
      //  [] __device__ (T& v1, T& v2) { return v1 + v2; },
      //  ans_operand[ind[2]], ans_operand[ind[1]]
      //);
      
      auto add3_t = cf.transform(
        ans_operand[ind[2]], ans_operand[ind[2]] + N, ans_operand[ind[1]],
        ans_operand[ind[0]],
        [] __device__ (T& v1, T& v2) { return v1 + v2; }
      );
  
      auto verify1_t = cf.kernel(
        32, 512, 0,
        verify<T>,
        operand[ind[0]], ans_operand[ind[0]], check, N
      );

      auto verify2_t = cf.kernel(
        32, 512, 0,
        verify<T>,
        operand[ind[1]], ans_operand[ind[1]], check, N
      );

      auto verify3_t = cf.kernel(
        32, 512, 0,
        verify<T>,
        operand[ind[2]], ans_operand[ind[2]], check, N
      );

      multi1_t.precede(add1_t);
      add1_t.precede(multi2_t);
      multi2_t.precede(add2_t);
      add2_t.precede(multi3_t);
      multi3_t.precede(add3_t);
      add3_t.precede(verify1_t).precede(verify2_t).precede(verify3_t);

      cf.offload();
      REQUIRE(*check);

    }).name("verify");

     //free memory
    auto deallocate_t = taskflow.emplace([&]() {
      for(int i = 0; i < 3; ++i) {
      REQUIRE(hipFree(operand[i]) == hipSuccess);
      REQUIRE(hipFree(ans_operand[i]) == hipSuccess);
      }

      REQUIRE(hipFree(check) == hipSuccess);
    }).name("deallocate");

    allocate_t.precede(initialize_t);
    initialize_t.precede(add_t);
    add_t.precede(verify_t);
    verify_t.precede(deallocate_t);

    executor.run(taskflow).wait();

  }

}

TEST_CASE("cudaFlowCapturer.rebind.kernel.int" * doctest::timeout(300)) {
  rebind_kernel<int>();
}

TEST_CASE("cudaFlowCapturer.rebind.kernel.float" * doctest::timeout(300)) {
  rebind_kernel<float>();
}

TEST_CASE("cudaFlowCapturer.rebind.kernel.double" * doctest::timeout(300)) {
  rebind_kernel<double>();
}

//----------------------------------------------------------------------
//rebind copy
//----------------------------------------------------------------------
template <typename T>
void rebind_copy() {
  tf::Executor executor;

  for(int N = 1; N < 65459; N = N * 2 + 1) {
    tf::Taskflow taskflow;

    std::vector<T> ha(N, N + 5);
    std::vector<T> hb(N, N - 31);
    std::vector<T> hc(N, N - 47);
    std::vector<T> hz(N);

    T* da {nullptr};
    T* db {nullptr};
    T* dc {nullptr};
    T* dz {nullptr};


    //allocate
    auto allocate_t = taskflow.emplace([&]() {
      REQUIRE(hipMalloc(&da, N * sizeof(T)) == hipSuccess);
      REQUIRE(hipMalloc(&db, N * sizeof(T)) == hipSuccess);
      REQUIRE(hipMalloc(&dc, N * sizeof(T)) == hipSuccess);
      REQUIRE(hipMalloc(&dz, N * sizeof(T)) == hipSuccess);
    }).name("allocate");


    //rebind_copy
    auto h2d_t = taskflow.emplace([&](tf::cudaFlowCapturer& cf) {
      auto h2d_t = cf.copy(da, ha.data(), N).name("h2d");
      cf.offload();

      cf.copy(h2d_t, db, hb.data(), N);
      cf.offload();

      cf.copy(h2d_t, dc, hc.data(), N);
      cf.offload();

    });

    auto kernel_t = taskflow.emplace([&](tf::cudaFlowCapturer& cf) {
      //auto add1_t = cf.transform(
      //  dz,  dz + N,
      //  [] __device__ (T& v1, T& v2) { return v1 + v2; },
      //  da, db
      //);
      
      auto add1_t = cf.transform(
        da, da+N, db,
        dz,
        [] __device__ (T& v1, T& v2) { return v1 + v2; }
      );

      //auto add2_t = cf.transform(
      //  dc,  dc + N,
      //  [] __device__ (T& v1, T& v2) { return v1 - v2; },
      //  dc, dz
      //);
      
      auto add2_t = cf.transform(
        dc, dc + N, dz,
        dc,
        [] __device__ (T& v1, T& v2) { return v1 - v2; }
      );

      add1_t.precede(add2_t);
    });

    auto d2h_t = taskflow.emplace([&](tf::cudaFlowCapturer& cf) {
      auto d2h_t = cf.copy(hc.data(), dc, N).name("d2h");
      cf.offload();

      cf.copy(d2h_t, hz.data(), dz, N);
      cf.offload();

    });

    //verify
    auto verify_t = taskflow.emplace([&]() {
      for(auto& c: hc) {
        REQUIRE(c == -21 - N);
      }

      for(auto& z: hz) {
        REQUIRE(z == 2 * N - 26);
      }
    });

     //free memory
    auto deallocate_t = taskflow.emplace([&]() {
      REQUIRE(hipFree(da) == hipSuccess);
      REQUIRE(hipFree(db) == hipSuccess);
      REQUIRE(hipFree(dc) == hipSuccess);
      REQUIRE(hipFree(dz) == hipSuccess);
    }).name("deallocate");

    allocate_t.precede(h2d_t);
    h2d_t.precede(kernel_t);
    kernel_t.precede(d2h_t);
    d2h_t.precede(verify_t);
    verify_t.precede(deallocate_t);

    executor.run(taskflow).wait();

  }
}

TEST_CASE("cudaFlowCapturer.rebind.copy.int" * doctest::timeout(300)) {
  rebind_copy<int>();
}

TEST_CASE("cudaFlowCapturer.rebind.copy.float" * doctest::timeout(300)) {
  rebind_copy<float>();
}

TEST_CASE("cudaFlowCapturer.rebind.copy.double" * doctest::timeout(300)) {
  rebind_copy<double>();
}


//----------------------------------------------------------------------
//rebind memcpy
//----------------------------------------------------------------------
template <typename T>
void rebind_memcpy() {
  tf::Executor executor;

  for(int N = 1; N < 65459; N = N * 2 + 1) {
    tf::Taskflow taskflow;

    std::vector<T> ha(N, N + 5);
    std::vector<T> hb(N, N - 31);
    std::vector<T> hc(N, N - 47);
    std::vector<T> hz(N);

    T* da {nullptr};
    T* db {nullptr};
    T* dc {nullptr};
    T* dz {nullptr};


    //allocate
    auto allocate_t = taskflow.emplace([&]() {
      REQUIRE(hipMalloc(&da, N * sizeof(T)) == hipSuccess);
      REQUIRE(hipMalloc(&db, N * sizeof(T)) == hipSuccess);
      REQUIRE(hipMalloc(&dc, N * sizeof(T)) == hipSuccess);
      REQUIRE(hipMalloc(&dz, N * sizeof(T)) == hipSuccess);
    }).name("allocate");


    //rebind_memcpy
    auto h2d_t = taskflow.emplace([&](tf::cudaFlowCapturer& cf) {
      auto h2d_t = cf.memcpy(da, ha.data(), sizeof(T) * N).name("h2d");
      cf.offload();

      cf.memcpy(h2d_t, db, hb.data(), sizeof(T) * N);
      cf.offload();

      cf.memcpy(h2d_t, dc, hc.data(), sizeof(T) * N);
      cf.offload();

    });

    auto kernel_t = taskflow.emplace([&](tf::cudaFlowCapturer& cf) {
      //auto add1_t = cf.transform(
      //  dz,  dz + N,
      //  [] __device__ (T& v1, T& v2) { return v1 + v2; },
      //  da, db
      //);
      
      auto add1_t = cf.transform(
        da, da + N, db,
        dz,
        [] __device__ (T& v1, T& v2) { return v1 + v2; }
      );

      //auto add2_t = cf.transform(
      //  dc,  dc + N,
      //  [] __device__ (T& v1, T& v2) { return v1 - v2; },
      //  dc, dz
      //);
      
      auto add2_t = cf.transform(
        dc, dc + N, dz,
        dc,
        [] __device__ (T& v1, T& v2) { return v1 - v2; }
      );

      add1_t.precede(add2_t);
    });

    auto d2h_t = taskflow.emplace([&](tf::cudaFlowCapturer& cf) {
      auto d2h_t = cf.memcpy(hc.data(), dc, sizeof(T) * N).name("d2h");
      cf.offload();

      cf.memcpy(d2h_t, hz.data(), dz, sizeof(T) * N);
      cf.offload();

    });

    //verify
    auto verify_t = taskflow.emplace([&]() {
      for(auto& c: hc) {
        REQUIRE(c == -21 - N);
      }

      for(auto& z: hz) {
        REQUIRE(z == 2 * N - 26);
      }
    });

     //free memory
    auto deallocate_t = taskflow.emplace([&]() {
      REQUIRE(hipFree(da) == hipSuccess);
      REQUIRE(hipFree(db) == hipSuccess);
      REQUIRE(hipFree(dc) == hipSuccess);
      REQUIRE(hipFree(dz) == hipSuccess);
    }).name("deallocate");

    allocate_t.precede(h2d_t);
    h2d_t.precede(kernel_t);
    kernel_t.precede(d2h_t);
    d2h_t.precede(verify_t);
    verify_t.precede(deallocate_t);

    executor.run(taskflow).wait();

  }
}

TEST_CASE("cudaFlowCapturer.rebind.memcpy.int" * doctest::timeout(300)) {
  rebind_memcpy<int>();
}

TEST_CASE("cudaFlowCapturer.rebind.memcpy.float" * doctest::timeout(300)) {
  rebind_memcpy<float>();
}

TEST_CASE("cudaFlowCapturer.rebind.memcpy.double" * doctest::timeout(300)) {
  rebind_memcpy<double>();
}

//----------------------------------------------------------------------
//rebind memset
//----------------------------------------------------------------------
template <typename T>
void rebind_memset() {

  tf::Executor executor;
  tf::Taskflow taskflow;

  for(size_t N = 1; N < 65199; N = N * 2 + 1) {

    taskflow.clear();

    T* a {nullptr};
    T* b {nullptr};

    T* ans_a {nullptr};
    T* ans_b {nullptr};
    
    bool* check {nullptr};

    //allocate
    auto allocate_t = taskflow.emplace([&]() {
      REQUIRE(hipMallocManaged(&a, N * sizeof(T)) == hipSuccess);
      REQUIRE(hipMallocManaged(&b, (N + 37) * sizeof(T)) == hipSuccess);

      REQUIRE(hipMallocManaged(&ans_a, N * sizeof(T)) == hipSuccess);
      REQUIRE(hipMallocManaged(&ans_b, (N + 37) * sizeof(T)) == hipSuccess);

      REQUIRE(hipMallocManaged(&check, sizeof(bool)) == hipSuccess);
    }).name("allocate");

    //initialize
    auto initialize_t = taskflow.emplace([&]() {
      std::generate_n(a, N, [&](){ return ::rand() % N - N / 2; });
      std::generate_n(b, N + 37, [&](){ return ::rand() % N + N / 2; });
      
      REQUIRE(hipMemset(ans_a, 0, N * sizeof(T)) == hipSuccess);
      REQUIRE(hipMemset(ans_b, 1, (N + 37) * sizeof(T)) == hipSuccess);

      *check = true;
    }).name("initialize"); 

    //rebind_memset
    auto memset_t = taskflow.emplace([&](tf::cudaFlowCapturer& cf) {
      auto memset_t = cf.memset(ans_a, 0, N * sizeof(T));
      cf.offload();

      cf.memset(memset_t, a, 0, N * sizeof(T));
      cf.offload();

      cf.memset(memset_t, b, 1, (N + 37) * sizeof(T));
      cf.offload();
    }).name("memset");

    //verify
    auto verify_t = taskflow.emplace([&](tf::cudaFlowCapturer& cf) {
      cf.kernel(
        32, 512, 0,
        verify<T>,
        a, ans_a, check, N
      );

      cf.kernel(
        32, 512, 0,
        verify<T>,
        b, ans_b, check, N + 37
      );

      cf.offload();
      REQUIRE(*check);
    }).name("verify");

    //free memory
    auto deallocate_t = taskflow.emplace([&]() {
      REQUIRE(hipFree(a) == hipSuccess);
      REQUIRE(hipFree(b) == hipSuccess);
      REQUIRE(hipFree(ans_a) == hipSuccess);
      REQUIRE(hipFree(ans_b) == hipSuccess);
      REQUIRE(hipFree(check) == hipSuccess);
    }).name("deallocate");

    allocate_t.precede(initialize_t);
    initialize_t.precede(memset_t);
    memset_t.precede(verify_t);
    verify_t.precede(deallocate_t);

    executor.run(taskflow).wait();
  }
}

TEST_CASE("cudaFlowCapturer.rebind.memset.int" * doctest::timeout(300)) {
  rebind_memset<int>();
}

TEST_CASE("cudaFlowCapturer.rebind.memset.float" * doctest::timeout(300)) {
  rebind_memset<float>();
}

TEST_CASE("cudaFlowCapturer.rebind.memset.double" * doctest::timeout(300)) {
  rebind_memset<double>();
}

// ----------------------------------------------------------------------------
// rebind algorithms
// ----------------------------------------------------------------------------

TEST_CASE("cudaFlowCapturer.rebind.algorithms") {

  tf::cudaFlowCapturer capturer;

  auto data = tf::cuda_malloc_shared<int>(10000);
  auto res = tf::cuda_malloc_shared<int>(1);

  auto task = capturer.for_each(
    data, data+10000, []__device__(int& i) {
      i = 10;
    }
  );

  capturer.offload();

  for(int i=0; i<10000; i++) {
    REQUIRE(data[i] == 10);
  }
  REQUIRE(capturer.num_tasks() == 1);
  
  // rebind to single task
  capturer.single_task(task, [=] __device__ () {*data = 2;});

  capturer.offload();
  
  REQUIRE(*data == 2);
  for(int i=1; i<10000; i++) {
    REQUIRE(data[i] == 10);
  }
  REQUIRE(capturer.num_tasks() == 1);
  
  // rebind to for each index
  capturer.for_each_index(task, 0, 10000, 1,
    [=] __device__ (int i) {
      data[i] = -23;
    }
  );

  capturer.offload();
  
  for(int i=0; i<10000; i++) {
    REQUIRE(data[i] == -23);
  }
  REQUIRE(capturer.num_tasks() == 1);

  // rebind to reduce
  *res = 10;
  capturer.reduce(task, data, data + 10000, res, 
    []__device__(int a, int b){ return a + b; }
  );

  capturer.offload();

  REQUIRE(*res == -229990);
  REQUIRE(capturer.num_tasks() == 1);
  
  // rebind to uninitialized reduce
  capturer.uninitialized_reduce(task, data, data + 10000, res, 
    []__device__(int a, int b){ return a + b; }
  );

  capturer.offload();

  REQUIRE(*res == -230000);
  REQUIRE(capturer.num_tasks() == 1);
  
  // rebind to single task
  capturer.single_task(task, [res]__device__(){ *res = 999; });
  REQUIRE(*res == -230000);

  capturer.offload();
  REQUIRE(*res == 999);
  REQUIRE(capturer.num_tasks() == 1);

  // clear the capturer
  capturer.clear();
  REQUIRE(capturer.num_tasks() == 0);

  capturer.offload();
  REQUIRE(*res == 999);
  for(int i=0; i<10000; i++) {
    REQUIRE(data[i] == -23);
  }

  // clear the memory
  tf::cuda_free(data);
  tf::cuda_free(res);
}
