#include <taskflow/cuda/cudaflow.hpp>
#include <taskflow/cuda/algorithm/merge.hpp>

int main(int argc, char* argv[]) {
  
  if(argc != 2) {
    std::cerr << "usage: ./cuda_merge N\n";
    std::exit(EXIT_FAILURE);
  }

  unsigned N = std::atoi(argv[1]);
  
  // gpu data
  auto da = tf::cuda_malloc_shared<int>(N);
  auto db = tf::cuda_malloc_shared<int>(N);
  auto dc = tf::cuda_malloc_shared<int>(N + N);

  // host data
  std::vector<int> ha(N), hb(N), hc(N + N);

  for(unsigned i=0; i<N; i++) {
    da[i] = ha[i] = rand()%100;
    db[i] = hb[i] = rand()%100;
  }
  
  std::sort(da, da+N);
  std::sort(db, db+N);
  std::sort(ha.begin(), ha.end());
  std::sort(hb.begin(), hb.end());

  // --------------------------------------------------------------------------
  // GPU merge
  // --------------------------------------------------------------------------

  auto beg = std::chrono::steady_clock::now();

  // allocate the buffer
  auto bufsz = tf::cuda_merge_buffer_size<tf::cudaDefaultExecutionPolicy>(N, N);
  tf::cudaDeviceVector<std::byte> buf(bufsz);

  tf::cuda_merge(tf::cudaDefaultExecutionPolicy{}, 
    da, da+N, db, db+N, dc, tf::cuda_less<int>{}, buf.data()
  );
  hipStreamSynchronize(0);
  auto end = std::chrono::steady_clock::now();

  std::cout << "GPU merge: " 
            << std::chrono::duration_cast<std::chrono::microseconds>(end-beg).count()
            << " us\n";
  
  // --------------------------------------------------------------------------
  // CPU merge
  // --------------------------------------------------------------------------
  beg = std::chrono::steady_clock::now();
  std::merge(ha.begin(), ha.end(), hb.begin(), hb.end(), hc.begin());
  end = std::chrono::steady_clock::now();
  
  std::cout << "CPU merge: " 
            << std::chrono::duration_cast<std::chrono::microseconds>(end-beg).count()
            << " us\n";

  // --------------------------------------------------------------------------
  // verify the result
  // --------------------------------------------------------------------------

  //for(unsigned i=0; i< N; i++) {
  //  printf("a[%u]=%d, b[%u]=%d\n", i, a[i], i, b[i]);
  //}
  //printf("\n");

  //for(unsigned i=0; i<N+N; i++) {
  //  printf("c[%u]=%d\n", i, c[i]);
  //}
  
  for(size_t i=0; i<N; i++) {
    if(dc[i] != hc[i]) {
      throw std::runtime_error("incorrect result");
    }
  }

  std::cout << "correct result\n";

  hipDeviceSynchronize();

};
