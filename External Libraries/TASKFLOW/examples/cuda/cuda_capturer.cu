#include "hip/hip_runtime.h"
// This program performs a simple single-precision Ax+Y operation
// using a cudaFlow capturer and verifies its result.

#include <taskflow/cuda/cudaflow.hpp>

// Kernel: saxpy
__global__ void saxpy(int n, float a, float *x, float *y) {
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) {
    y[i] = a*x[i] + y[i];
  }
}

// Function: main
int main() {
  
  const unsigned N = 1<<20;

  tf::Taskflow taskflow ("saxpy-flow");
  tf::Executor executor;

  std::vector<float> hx(N, 1.0f), hy(N, 2.0f);

  auto dx = tf::cuda_malloc_device<float>(N);
  auto dy = tf::cuda_malloc_device<float>(N);

  taskflow.emplace([&](tf::cudaFlowCapturer& cf) {
    std::cout << "running cudaflow capturer ...\n";
    auto h2d_x = cf.copy(dx, hx.data(), N).name("h2d_x");
    auto h2d_y = cf.copy(dy, hy.data(), N).name("h2d_y");
    auto d2h_x = cf.copy(hx.data(), dx, N).name("d2h_x");
    auto d2h_y = cf.copy(hy.data(), dy, N).name("d2h_y");
    auto kernel = cf.kernel((N+255)/256, 256, 0, saxpy, N, 2.0f, dx, dy)
                    .name("saxpy");
    kernel.succeed(h2d_x, h2d_y)
          .precede(d2h_x, d2h_y);
  }).name("saxpy");
  
  executor.run(taskflow).wait();

  float max_error = 0.0f;
  for (size_t i = 0; i < N; i++) {
    max_error = std::max(max_error, abs(hx[i]-1.0f));
    max_error = std::max(max_error, abs(hy[i]-4.0f));
  }
  std::cout << "saxpy finished with max error: " << max_error << '\n';

  // free memory
  tf::cuda_free(dx);
  tf::cuda_free(dy);

  std::cout << "dumping the taskflow ...\n";
  taskflow.dump(std::cout);

  return 0;
}

